#include "hip/hip_runtime.h"
#include <iostream>
#include <cstdlib>
#include <cmath>
#include <vector>

using namespace std;

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <string.h>

#include <hip/hip_runtime_api.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

#include <SDL/SDL.h>
#include <SDL/SDL_image.h>

#include "fblib.h"
#include "imgload.h"
#include "inputs.h"

#include "globals.h"

class Site
{
	public:
		float logis[NVAR], targ[NVAR];
		int bits[NVAR];
		float fixed[2];
		float W;
};

__device__ float logistic(float x)
{
	return fmaxf(0.0001f,fminf(0.9999f,LOGISTIC_R * x * (1.0f - x)));
	//~ return x;
}

Site *hostGrid, *devGrid;

__global__ void iterateLogistic(Site *devGrid)
{
	long int idx = blockIdx.x*blockDim.x + threadIdx.x;
   
	if (idx<NSITES*NVAR)
	{   
		int site = idx/NVAR;
		int var = idx%NVAR;
		
		for (int i=0;i<NITER;i++)
		{
			devGrid[site].logis[var] = logistic( devGrid[site].logis[var] );
		}
		
		float d1 = abs(devGrid[site].logis[var] - devGrid[site].fixed[0]);
		float d2 = abs(devGrid[site].logis[var] - devGrid[site].fixed[1]);
		
		devGrid[site].bits[var] = (d2<d1);
	}
}

__global__ void getFixed(Site *devGrid)
{
	long int idx = blockIdx.x*blockDim.x + threadIdx.x;
	
	int rseq[NVAR] =
	{
		1,0,1,
		1,0,0,
		1,1,0,
		1,0,0,
		1,0,0,
		1,0,0,
		1,0,0,
		1,0,0,
		1,0,0,
		1,0,0,
		1,0,0,
		1,0,0
	};
	
	if (idx<NSITES*3)
	{   
		int fid = idx/NSITES;
		int sid = idx%NSITES;
		int result = 0;
		
		int bidx = fid*NBITS;
		float dist = 0;
				
		for (int i=0;i<NBITS;i++)
		{
			result += (1<<i)*(devGrid[sid].bits[bidx] ^ rseq[bidx]);
			dist += (devGrid[sid].bits[bidx] != (TARGET>>i)%2);
			bidx++;
		}
		
		if (fid<2)
		{
			devGrid[sid].fixed[fid] = ((float)result)/(float)(1<<NBITS);
		}
		else
		{
			devGrid[sid].W = WBIAS * fmaxf(0.0f,((1.0f-dist/(float)NBITS) - 0.5f));//WBIAS * ( (1.0-dist/(float)NBITS) - 0.5 );
		}
	}
}

__global__ void Contract(Site *devGrid)
{
	long int idx = blockIdx.x*blockDim.x + threadIdx.x;
   
	if (idx<NSITES*NVAR)
	{   
		int site = idx/NVAR;
		int var = idx%NVAR;

		// Determine target point based on neighbors
		devGrid[site].targ[var] = devGrid[site].fixed[ devGrid[site].bits[var] ];
		float norm = 1.0f;
		
		for (int j=0;j<2;j++)
		{
			int k=2*j-1 + site;
			
			if (k<0) k += NSITES;
			if (k>=NSITES) k -= NSITES;
			
			norm += devGrid[k].W;
			devGrid[site].targ[var] += devGrid[k].fixed[ devGrid[k].bits[var] ] * devGrid[k].W;			
		}

		devGrid[site].targ[var] /= norm;

		//~ if (devGrid[site].W > devGrid[(site+1)%NSITES].W)
			//~ devGrid[site].targ[var] = devGrid[site].fixed[ devGrid[site].bits[var] ];
		//~ else
			//~ devGrid[site].targ[var] = devGrid[(site+1)%NSITES].fixed[ devGrid[(site+1)%NSITES].bits[var] ];
		devGrid[site].logis[var] += CRATE*(devGrid[site].targ[var] - devGrid[site].logis[var]);
	}
}

void Iterate()
{
	int block_size = BSIZE;
	int n_blocks1 = NSITES*NVAR/block_size + (NSITES*NVAR%block_size == 0 ? 0 : 1);  
	int n_blocks2 = (NSITES*3)/block_size + ((NSITES*3)%block_size == 0 ? 0 : 1);  
	
	iterateLogistic <<< n_blocks1, block_size >>> (devGrid);
	getFixed <<< n_blocks2, block_size >>> (devGrid);
	Contract <<< n_blocks1, block_size >>> (devGrid);
	
	hipMemcpy(hostGrid, devGrid, sizeof(Site)*NSITES, hipMemcpyDeviceToHost);
}

void Init()
{
	hostGrid=(Site*)malloc(sizeof(Site)*NSITES);
	hipMalloc((void**)&devGrid, NSITES*sizeof(Site));
	
	for (int i=0;i<NSITES;i++)
	{
		hostGrid[i].fixed[0] = 0.25;
		hostGrid[i].fixed[1] = 0.75;
		for (int j=0;j<NVAR;j++)
			hostGrid[i].logis[j] = (rand()%1000000)/1000000.0;			
		//hostGrid[i].W = (rand()%100001)/100000.0;
	}
	
	hipMemcpy(devGrid, hostGrid, sizeof(Site)*NSITES, hipMemcpyHostToDevice);
	
	XRes = WIDTH; YRes = HEIGHT; Bpp = 3;
	InitSDL();
	ScreenBuf=(unsigned char*)malloc(XRes*YRes*Bpp);
	memset(ScreenBuf,0,XRes*YRes*Bpp);
}

int iter = 0;

void Render()
{
	int y = iter%NSITES;
	
	for (int x=0;x<NSITES;x++)
	{
		int r,g,b;
		float w = (hostGrid[x].W/WBIAS+0.5);
		
		g=192-92*w;
		r=255*(1.0-w);
		b=255*w;
		
		if (w != w) printf("NaN!\n");
		//if (w<0) r=g=b=255;
		
		if (r<0) r=0; if (r>255) r=255;
		if (g<0) g=0; if (g>255) g=255;
		if (b<0) b=0; if (b>255) b=255;
		
		for (int y2=0;y2<CELLSIZE;y2++)
		{
			for (int x2=0;x2<CELLSIZE;x2++)
			{
				int xm = CELLSIZE*x+x2;
				int ym = CELLSIZE*y+y2;
				ScreenBuf[(xm+ym*XRes)*Bpp]=b;
				ScreenBuf[(xm+ym*XRes)*Bpp+1]=g;
				ScreenBuf[(xm+ym*XRes)*Bpp+2]=r;
			}
		}
	}
}

int main(int argc, char **argv)
{
	Init();
	
	while (1)
	{
		int Ch=ReadKey();
		
		if (Ch=='q') return 0;

		Iterate();
		Render();
		BlitBuf(ScreenBuf,0,0,WIDTH,HEIGHT);
		iter++;
		
		double fbar = 0;
		
		for (int i=0;i<NSITES;i++)
		{
			fbar += (hostGrid[i].W/WBIAS+0.5);
		}
		fbar /= (double)NSITES;
		
		if (iter%100==0)
			printf("%.6g\n",fbar);
	}
}
